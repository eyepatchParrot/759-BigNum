#include "hip/hip_runtime.h"
// Notes at bottom.
#include <iostream>
#include <string>
#include <limits>

#include <gmp.h>
#include <gmpxx.h>

#include "int.h"
#include "cksum.h"

uint16_t mpz_cksum(const mpz_class& n) {
  auto p_n = n.get_mpz_t();
  return cksum((uint8_t*)mpz_limbs_read(p_n), mpz_size(p_n) * sizeof(mp_limb_t));
}

int main() {
  using GPU::Int;

  // *** test loads and cksum
  for (int i = 0; i < 0xFFFF; i += 0x10) {
    mpz_class mpz_i = i;
    Int gpu_i(mpz_i);
    if (mpz_cksum(mpz_i) != gpu_i.cksum()) {
      std::string mp_str = mpz_i.get_str(16), d_str = gpu_i;
      if (d_str == mp_str) continue;
      std::cerr << mp_str << '\n' << d_str << '\n';
      assert(mp_str == d_str);
    }
  }
  std::cout << "PASS load inc by 16\n";

  mpz_class mpz_i = std::numeric_limits<long>::max();
  for (int i = 0; i < 100; i++, mpz_i += mpz_i) {
    Int gpu_i = mpz_i;

    if (mpz_cksum(mpz_i) != gpu_i.cksum()) {
      std::string d_str = gpu_i, mp_str = mpz_i.get_str(16);
      if (d_str == mp_str) continue;
      std::cerr << mp_str << '\n' << d_str << '\n';
      assert(mp_str == d_str);
    }
  }
  std::cout << "PASS load sum by itself\n";

  for (int i = 0; i < 10; i++, mpz_i *= mpz_i) {
    Int gpu_i = mpz_i;

    if (mpz_cksum(mpz_i) != gpu_i.cksum()) {
      std::string d_str = gpu_i, mp_str = mpz_i.get_str(16);
      if (d_str == mp_str) continue;
      std::cerr << mp_str << '\n' << d_str << '\n';
      assert(mp_str == d_str);
    }
  }

  std::cout << "PASS load squared\n";

  /*
   * test addition without carries
   */
  // single limb
  mpz_class limb_n = 1_mpz << 64;
  assert(mpz_size(limb_n.get_mpz_t()) == 2);


#define GPU_OK(N1, OP, N2, NAME) \
  do { \
    mpz_class t1 = N1, t2 = N2; \
    Int t3 = Int(t1) OP Int(t2); \
    if (mpz_cksum(t1 OP t2) != t3.cksum()) { \
      mpz_class r = t1 OP t2; \
        std::cerr << r.get_str(16) << '\n' << std::string(t3) << '\n'; \
        assert(!NAME); \
    } else { \
      std::cout << "PASS " << NAME << '\n'; \
    } \
  } while (0)

  GPU_OK(1_mpz, +, 1_mpz, "1+1=2");
  GPU_OK(limb_n, +, limb_n, "10 + 10 = 20");
  GPU_OK(limb_n + 1_mpz, +, limb_n + 1_mpz, "11 + 11 = 22");
  GPU_OK((limb_n - 1) * limb_n, +, (limb_n - 1), "F0 + 0F = FF");

  // carry tests
  GPU_OK(limb_n - 1, +, limb_n - 1, "F + F = 1E");
  GPU_OK(limb_n - 1, +, 1, "F + 1 = 10");
  GPU_OK((limb_n - 1) * limb_n, +, (limb_n - 1) * limb_n, "F0 + F0 = 1E0");
  GPU_OK((limb_n * limb_n - 1), +, (limb_n * limb_n - 1), "FF + FF = 1FE");
  GPU_OK((limb_n * limb_n - 1), +, 1_mpz, "FF + 1 = 100");
  GPU_OK(limb_n * (limb_n - 2) + limb_n - 1, +, limb_n + 1, "EF + 11 = 100");

  // carry block tests
  mpz_class block_n = 1_mpz << (64*256);
  assert(mpz_size(block_n.get_mpz_t()) == 257);
  GPU_OK(block_n, +, block_n, "BLK 10 + 10 = 20");
  GPU_OK(block_n - 1, +, block_n - 1, "BLK F + F = 1E");
  GPU_OK(block_n - 1, +, 1, "BLK F + 1 = 10");
  GPU_OK((block_n - 1) * block_n, +, (block_n - 1) * block_n, "BLK F0 + F0 = 1E0");
  GPU_OK((block_n * block_n - 1), +, (block_n * block_n - 1), "BLK FF + FF = 1FE");
  GPU_OK((block_n * block_n - 1), +, 1_mpz, "BLK FF + 1 = 100");
  GPU_OK(block_n * (block_n - 2) + block_n - 1, +, block_n + 1, "BLK EF + 11 = 100");

  // times tests
  GPU_OK(1_mpz, *, 1_mpz, "1 * 1 = 1");
  GPU_OK(limb_n, *, 1_mpz, "10 * 1 = 10");
  GPU_OK(block_n, *, 1_mpz, "BLK 10 * 1 = 10");
  GPU_OK(limb_n - 1, *, 2_mpz, "F * 2 = 1E");
  GPU_OK((limb_n - 1) * limb_n, *, 2_mpz, "F0 * 2 = 1E0");
  GPU_OK((limb_n * limb_n - 1), *, 2_mpz, "FF * 2 = 1FE");
  GPU_OK(block_n, *, 2_mpz, "BLK 10 * 2 = 20");
  GPU_OK(block_n - 1, *, 2_mpz, "BLK F * 2 = 1E");
  GPU_OK(limb_n - 1, *, limb_n - 1, "F * F = FFFE0001");
  GPU_OK(block_n - 1, *, block_n - 1, "BLK F * F = FFFE0001");

  // TODO randomized testing


  return 0;
}
