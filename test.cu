#include "hip/hip_runtime.h"
// Notes at bottom.
#include <iostream>
#include <string>
#include <limits>

#include <gmp.h>
#include <gmpxx.h>

#include "int.h"
#include "cksum.h"

uint16_t mpz_cksum(const mpz_class& n) {
  auto p_n = n.get_mpz_t();
  return cksum((uint8_t*)mpz_limbs_read(p_n), mpz_size(p_n) * sizeof(mp_limb_t));
}

int main() {
  using GPU::Int;

  // *** test loads and cksum
  for (int i = 0; i < 0xFFFF; i += 0x10) {
    mpz_class mpz_i = i;
    Int gpu_i(mpz_i);
    if (mpz_cksum(mpz_i) != gpu_i.cksum()) {
      std::string mp_str = mpz_i.get_str(16), d_str = gpu_i;
      if (d_str == mp_str) continue;
      std::cerr << mp_str << '\n' << d_str << '\n';
      assert(mp_str == d_str);
    }
  }
  std::cout << "PASS load inc by 16\n";

  mpz_class mpz_i = std::numeric_limits<long>::max();
  for (int i = 0; i < 100; i++, mpz_i += mpz_i) {
    Int gpu_i = mpz_i;

    if (mpz_cksum(mpz_i) != gpu_i.cksum()) {
      std::string d_str = gpu_i, mp_str = mpz_i.get_str(16);
      if (d_str == mp_str) continue;
      std::cerr << mp_str << '\n' << d_str << '\n';
      assert(mp_str == d_str);
    }
  }
  std::cout << "PASS load sum by itself\n";

  for (int i = 0; i < 10; i++, mpz_i *= mpz_i) {
    Int gpu_i = mpz_i;

    if (mpz_cksum(mpz_i) != gpu_i.cksum()) {
      std::string d_str = gpu_i, mp_str = mpz_i.get_str(16);
      if (d_str == mp_str) continue;
      std::cerr << mp_str << '\n' << d_str << '\n';
      assert(mp_str == d_str);
    }
  }

  std::cout << "PASS load squared\n";

  /*
   * test addition without carries
   */
  // single limb
  mpz_class limb_n = 1_mpz << 64;
  assert(mpz_size(limb_n.get_mpz_t()) == 2);

  mpz_class block_n = 1_mpz << (64*256);
  assert(mpz_size(block_n.get_mpz_t()) == 257);

  gmp_randstate_t rands;
  gmp_randinit_default(rands);
  gmp_randseed_ui(rands, 42);

  const int n_tests = 10;

#define GPU_OK(N1, OP, N2, NAME) \
  do { \
    mpz_class t1 = N1, t2 = N2; \
    Int t3 = Int(t1) OP Int(t2); \
    if (mpz_cksum(t1 OP t2) != t3.cksum()) { \
      mpz_class r = t1 OP t2; \
        std::cerr << "FAIL " << NAME << '\n' << t1.get_str(16) << '\n' << t2.get_str(16) << '\n' << r.get_str(16) << '\n' << std::string(t3) << '\n'; \
        assert(!NAME); \
    } else { \
      if ("" != NAME) std::cout << "PASS " << NAME << '\n'; \
    } \
  } while (0)

  // addition tests
  // zero
  GPU_OK(1_mpz, +, 0_mpz, "1+0=1");
  GPU_OK(limb_n - 1, +, 0_mpz, "F+0=F");
  GPU_OK(limb_n * (limb_n - 1), +, 0_mpz, "F0+0=F0");
  GPU_OK(block_n - 1, +, 0_mpz, "BLK F + 0 = F");
  GPU_OK(block_n, +, 0_mpz, "BLK 10 + 0 = 10");

  GPU_OK(1_mpz, +, 1_mpz, "1+1=2");
  GPU_OK(limb_n, +, limb_n, "10 + 10 = 20");
  GPU_OK(limb_n + 1_mpz, +, limb_n + 1_mpz, "11 + 11 = 22");
  GPU_OK((limb_n - 1) * limb_n, +, (limb_n - 1), "F0 + 0F = FF");

  // carry tests
  GPU_OK(limb_n - 1, +, limb_n - 1, "F + F = 1E");
  GPU_OK(limb_n - 1, +, 1, "F + 1 = 10");
  GPU_OK((limb_n - 1) * limb_n, +, (limb_n - 1) * limb_n, "F0 + F0 = 1E0");
  GPU_OK((limb_n * limb_n - 1), +, (limb_n * limb_n - 1), "FF + FF = 1FE");
  GPU_OK((limb_n * limb_n - 1), +, 1_mpz, "FF + 1 = 100");
  GPU_OK(limb_n * (limb_n - 2) + limb_n - 1, +, limb_n + 1, "EF + 11 = 100");

  // carry block tests
  GPU_OK(block_n, +, block_n, "BLK 10 + 10 = 20");
  GPU_OK(block_n - 1, +, block_n - 1, "BLK F + F = 1E");
  GPU_OK(block_n - 1, +, 1, "BLK F + 1 = 10");
  GPU_OK((block_n - 1) * block_n, +, (block_n - 1) * block_n, "BLK F0 + F0 = 1E0");
  GPU_OK((block_n * block_n - 1), +, (block_n * block_n - 1), "BLK FF + FF = 1FE");
  GPU_OK((block_n * block_n - 1), +, 1_mpz, "BLK FF + 1 = 100");
  GPU_OK(block_n * (block_n - 2) + block_n - 1, +, block_n + 1, "BLK EF + 11 = 100");

  // random addition tests
  //for (int sz = 1; sz < std::numeric_limits<int>::max() / 2; sz *= 2) {
  // TODO figure out 1048576
  for (int sz = 64; sz <= 1048576; sz *= 2) {
    mpz_class a,b;
    for (int i = 0; i < n_tests; i++) {
      mpz_rrandomb(a.get_mpz_t(), rands, sz);
      mpz_rrandomb(b.get_mpz_t(), rands, sz);
      GPU_OK(a, +, b, "");
    }
  }
  GPU_OK(1,+,1,"RAND +");

  // times tests
  // zero
  GPU_OK(1_mpz, *, 0_mpz, "1*0=0");
  GPU_OK(limb_n - 1, *, 0_mpz, "F*0=0");
  GPU_OK(limb_n * (limb_n - 1), *, 0_mpz, "F0*0=0");
  GPU_OK(block_n - 1, *, 0_mpz, "BLK F * 0 = 0");
  GPU_OK(block_n, *, 0_mpz, "BLK 10 * 0 = 0");

  GPU_OK(1_mpz, *, 1_mpz, "1 * 1 = 1");
  GPU_OK(limb_n, *, 1_mpz, "10 * 1 = 10");
  GPU_OK(block_n, *, 1_mpz, "BLK 10 * 1 = 10");
  GPU_OK(limb_n - 1, *, 2_mpz, "F * 2 = 1E");
  GPU_OK((limb_n - 1) * limb_n, *, 2_mpz, "F0 * 2 = 1E0");
  GPU_OK((limb_n * limb_n - 1), *, 2_mpz, "FF * 2 = 1FE");
  GPU_OK(block_n, *, 2_mpz, "BLK 10 * 2 = 20");
  GPU_OK(block_n - 1, *, 2_mpz, "BLK F * 2 = 1E");
  GPU_OK(limb_n - 1, *, limb_n - 1, "F * F = FFFE0001");
  GPU_OK(block_n - 1, *, block_n - 1, "BLK F * F = FFFE0001");

  // random * test
  for (int sz = 1; sz <= 2048 << (2*5); sz *= 2) {
    mpz_class a,b;
    for (int i = 0; i < n_tests; i++) {
      mpz_rrandomb(a.get_mpz_t(), rands, sz);
      mpz_rrandomb(b.get_mpz_t(), rands, sz);
      GPU_OK(a, *, b, "");
      //GPU_OK(a, *, b, (std::to_string(sz) + " " + std::to_string(i)).c_str());
    }
  }
  GPU_OK(1,*,1,"RAND *");

  return 0;
}
